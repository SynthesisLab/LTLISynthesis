#include "hip/hip_runtime.h"
#include <set>
#include <vector>
#include <chrono>
#include <fstream>
#include <iostream>
#include <thrust/copy.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include <warpcore/hash_set.cuh>

using UINT_64 = std::uint64_t;

const std::size_t maxNumOfTraces = 64;

__constant__ char d_traceLen[maxNumOfTraces];

inline
hipError_t checkCuda(hipError_t res) {
#ifndef MEASUREMENT_MODE
    if (res != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(res));
        assert(res == hipSuccess);
    }
#endif
    return res;
}

// Finding the left and right indices that makes the final LTL to bring to the host later
__global__ void generateResIndices(
    const int index,
    const int alphabetSize,
    const int* d_leftIdx,
    const int* d_rightIdx,
    int* d_FinalLTLIdx)
{

    int resIdx = 0;
    while (d_FinalLTLIdx[resIdx] != -1) resIdx++;
    int queue[600];
    queue[0] = index;
    int head = 0;
    int tail = 1;
    while (head < tail) {
        int ltl = queue[head];
        int l = d_leftIdx[ltl];
        int r = d_rightIdx[ltl];
        d_FinalLTLIdx[resIdx++] = ltl;
        d_FinalLTLIdx[resIdx++] = l;
        d_FinalLTLIdx[resIdx++] = r;
        if (l >= alphabetSize) queue[tail++] = l;
        if (r >= alphabetSize) queue[tail++] = r;
        head++;
    }

}

__device__ void makeRlxUnqChkCSs(
    UINT_64* CS,
    UINT_64& hCS,
    UINT_64& lCS,
    const int numOfTraces,
    const int RlxUnqChkTyp,
    const int lenSum)
{

    if (lenSum > 126) {

        // we need an relaxed uniqueness check

        switch (RlxUnqChkTyp) {

        case 1: {

            const int stride = lenSum / 126;

            int j = 0;
            for (int i = 0; i < numOfTraces; ++i) {
                for (int k = 0; k < d_traceLen[i]; k += stride, ++j) {
                    if (j < 63) {
                        if (CS[i] & ((UINT_64)1 << k)) lCS |= (UINT_64)1 << j;
                    }
                    else if (j < 126) {
                        if (CS[i] & ((UINT_64)1 << k)) hCS |= (UINT_64)1 << (j - 63);
                    }
                    else break;
                }
            }

            break;
        }

        case 2: {

            int j = 0;
            for (int i = 0; i < numOfTraces; ++i) {
                UINT_64 bitPtr = 1;
                int maxbitsForThisTrace = (126 * d_traceLen[i] + lenSum) / lenSum;
                for (int k = 0; k < maxbitsForThisTrace; ++k, ++j, bitPtr <<= 1) {
                    if (j < 63) {
                        if (CS[i] & bitPtr) lCS |= (UINT_64)1 << j;
                    }
                    else if (j < 126) {
                        if (CS[i] & bitPtr) hCS |= (UINT_64)1 << (j - 63);
                    }
                    else break;
                }
            }

            break;
        }

        case 3: {

            for (int i = 0; i < numOfTraces; ++i) {
                UINT_64 x = CS[i];
                x = (x ^ (x >> 30)) * UINT64_C(0xbf58476d1ce4e5b9);
                x = (x ^ (x >> 27)) * UINT64_C(0x94d049bb133111eb);
                x = x ^ (x >> 31);
                if (i < numOfTraces / 2) hCS ^= x; else lCS ^= x;
            }

            break;
        }

        }

    }
    else {

        // the result will be minimal

        int j = 0;
        for (int i = 0; i < numOfTraces; ++i) {
            UINT_64 bitPtr = 1;
            for (int k = 0; k < d_traceLen[i]; ++k, ++j, bitPtr <<= 1) {
                if (j < 63) {
                    if (CS[i] & bitPtr) lCS |= (UINT_64)1 << j;
                }
                else if (j < 126) {
                    if (CS[i] & bitPtr) hCS |= (UINT_64)1 << (j - 63);
                }
                else break;
            }
        }

    }

}

// Initialising the hashSets with the alphabet before starting the enumeration
template<class hash_set_t>
__global__ void hashSetsInitialisation(
    const int numOfTraces,
    const int RlxUnqChkTyp,
    const int lenSum,
    hash_set_t cHashSet, hash_set_t iHashSet,
    UINT_64* d_LTLcache)
{

    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    UINT_64 CS[maxNumOfTraces];

    for (int i = 0; i < numOfTraces; ++i)
        CS[i] = d_LTLcache[tid * numOfTraces + i];

    UINT_64 hCS{}, lCS{};
    makeRlxUnqChkCSs(CS, hCS, lCS, numOfTraces, RlxUnqChkTyp, lenSum);

    const auto group = warpcore::cg::tiled_partition<1>(warpcore::cg::this_thread_block());
    int H = cHashSet.insert(hCS, group); int L = cHashSet.insert(lCS, group);
    H = (H > 0) ? H : -H; L = (L > 0) ? L : -L;
    UINT_64 HL = H; HL <<= 32; HL |= L;
    iHashSet.insert(HL, group);

}

__device__ void applyOperator(
    char op,
    UINT_64* CS,
    UINT_64* d_LTLcache,
    int ldx, int rdx,
    int numOfTraces)
{

    switch (op) {
    case '~': {
        for (int i = 0; i < numOfTraces; ++i) {
            UINT_64 negationFixer = ((UINT_64)1 << d_traceLen[i]) - 1;
            CS[i] = ~d_LTLcache[ldx * numOfTraces + i] & negationFixer;
        }
        break;
    }
    case '&': {
        for (int i = 0; i < numOfTraces; ++i) {
            CS[i] = d_LTLcache[ldx * numOfTraces + i] & d_LTLcache[rdx * numOfTraces + i];
        }
        break;
    }
    case '|': {
        for (int i = 0; i < numOfTraces; ++i) {
            CS[i] = d_LTLcache[ldx * numOfTraces + i] | d_LTLcache[rdx * numOfTraces + i];
        }
        break;
    }
    case 'X': {
        for (int i = 0; i < numOfTraces; ++i) {
            CS[i] = d_LTLcache[ldx * numOfTraces + i] >> 1;
        }
        break;
    }
    case 'F': {
        for (int i = 0; i < numOfTraces; ++i) {
            CS[i] = d_LTLcache[ldx * numOfTraces + i];
            CS[i] |= CS[i] >> 1; CS[i] |= CS[i] >> 2; CS[i] |= CS[i] >> 4;
            CS[i] |= CS[i] >> 8; CS[i] |= CS[i] >> 16; CS[i] |= CS[i] >> 32;
        }
        break;
    }
    case 'G': {
        for (int i = 0; i < numOfTraces; ++i) {
            CS[i] = d_LTLcache[ldx * numOfTraces + i];
            UINT_64 cs = ~CS[i] & (((UINT_64)1 << d_traceLen[i]) - 1);
            cs |= cs >> 1; cs |= cs >> 2; cs |= cs >> 4;
            cs |= cs >> 8; cs |= cs >> 16; cs |= cs >> 32;
            CS[i] &= ~cs;
        }
        break;
    }
    case 'U': {
        for (int i = 0; i < numOfTraces; ++i) {
            UINT_64 l = d_LTLcache[ldx * numOfTraces + i];
            UINT_64 r = d_LTLcache[rdx * numOfTraces + i];
            r |= l & (r >> 1);  l &= l >> 1;
            r |= l & (r >> 2);  l &= l >> 2;
            r |= l & (r >> 4);  l &= l >> 4;
            r |= l & (r >> 8);  l &= l >> 8;
            r |= l & (r >> 16); l &= l >> 16;
            r |= l & (r >> 32);
            CS[i] = r;
        }
        break;
    }
    default:
        printf("Opérateur non pris en charge: %c\n", op);
        break;
    }

}

__device__ void processOnTheFly(
    UINT_64* CS,
    int tid,
    int numOfTraces, int numOfP,
    int ldx, int rdx,
    int* d_temp_leftIdx, int* d_temp_rightIdx,
    int* d_FinalLTLIdx)
{

    bool found = true;
    for (int i = 0; found && i < numOfP; ++i) if (!(CS[i] & 1)) found = false;
    for (int i = numOfP; found && i < numOfTraces; ++i) if (CS[i] & 1) found = false;

    if (found) {
        d_temp_leftIdx[tid] = ldx; d_temp_rightIdx[tid] = rdx;
        atomicCAS(d_FinalLTLIdx, -1, tid);
    }

}

template <typename hash_set_t>
__device__ bool processUniqueCS(
    UINT_64* CS,
    const int numOfTraces,
    const int RlxUnqChkTyp,
    const int lenSum,
    hash_set_t& cHashSet, hash_set_t& iHashSet)
{

    UINT_64 hCS{}, lCS{};
    makeRlxUnqChkCSs(CS, hCS, lCS, numOfTraces, RlxUnqChkTyp, lenSum);

    const auto group = warpcore::cg::tiled_partition<1>(warpcore::cg::this_thread_block());
    int H = cHashSet.insert(hCS, group); int L = cHashSet.insert(lCS, group);
    H = (H > 0) ? H : -H; L = (L > 0) ? L : -L;
    UINT_64 HL = H; HL <<= 32; HL |= L;
    return (iHashSet.insert(HL, group) > 0) ? false : true;

}

__device__ void insertInCache(
    bool CS_is_unique,
    UINT_64* CS,
    int tid,
    int numOfTraces, int numOfP,
    int ldx, int rdx,
    UINT_64* d_temp_LTLcache,
    int* d_temp_leftIdx, int* d_temp_rightIdx,
    int* d_FinalLTLIdx)
{

    if (CS_is_unique) {

        for (int i = 0; i < numOfTraces; ++i)
            d_temp_LTLcache[tid * numOfTraces + i] = CS[i];
        d_temp_leftIdx[tid] = ldx; d_temp_rightIdx[tid] = rdx;

        bool found = true;
        for (int i = 0; found && i < numOfP; ++i) if (!(CS[i] & 1)) found = false;
        for (int i = numOfP; found && i < numOfTraces; ++i) if (CS[i] & 1) found = false;
        if (found) atomicCAS(d_FinalLTLIdx, -1, tid);

    }
    else {

        for (int i = 0; i < numOfTraces; ++i)
            d_temp_LTLcache[tid * numOfTraces + i] = (UINT_64)-1;
        d_temp_leftIdx[tid] = -1; d_temp_rightIdx[tid] = -1;

    }

}

template<class hash_set_t>
__global__ void processOperator(
    const char op,
    const int idx1, const int idx2,
    const int idx3, const int idx4,
    const int numOfP, const int numOfN,
    const int RlxUnqChkTyp,
    const int lenSum,
    const bool onTheFly,
    UINT_64* d_LTLcache, UINT_64* d_temp_LTLcache,
    int* d_temp_leftIdx, int* d_temp_rightIdx,
    hash_set_t cHashSet, hash_set_t iHashSet,
    int* d_FinalLTLIdx)
{

    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int numOfTraces = numOfP + numOfN;
    bool isUnary = (op == '~' || op == 'X' || op == 'F' || op == 'G');
    int maxTid = isUnary ? (idx2 - idx1 + 1) : ((idx4 - idx3 + 1) * (idx2 - idx1 + 1));

    if (tid < maxTid) {

        int ldx = isUnary ? (idx1 + tid) : (idx1 + tid / (idx4 - idx3 + 1));
        int rdx = isUnary ? 0 : (idx3 + tid % (idx4 - idx3 + 1));

        if (op == 'U') {

            UINT_64 CS1[maxNumOfTraces], CS2[maxNumOfTraces];
            applyOperator('U', CS1, d_LTLcache, ldx, rdx, numOfTraces);
            applyOperator('U', CS2, d_LTLcache, rdx, ldx, numOfTraces);

            if (onTheFly) {
                processOnTheFly(
                    CS1, tid * 2, numOfTraces, numOfP, ldx, rdx,
                    d_temp_leftIdx, d_temp_rightIdx, d_FinalLTLIdx
                );
                processOnTheFly(
                    CS2, tid * 2 + 1, numOfTraces, numOfP, rdx, ldx,
                    d_temp_leftIdx, d_temp_rightIdx, d_FinalLTLIdx
                );
            }

            else {
                bool CS1_is_unique =
                    processUniqueCS(CS1, numOfTraces, RlxUnqChkTyp, lenSum, cHashSet, iHashSet);
                bool CS2_is_unique =
                    processUniqueCS(CS2, numOfTraces, RlxUnqChkTyp, lenSum, cHashSet, iHashSet);
                insertInCache(
                    CS1_is_unique, CS1, tid * 2, numOfTraces, numOfP, ldx, rdx,
                    d_temp_LTLcache, d_temp_leftIdx, d_temp_rightIdx, d_FinalLTLIdx
                );
                insertInCache(
                    CS2_is_unique, CS2, tid * 2 + 1, numOfTraces, numOfP, rdx, ldx,
                    d_temp_LTLcache, d_temp_leftIdx, d_temp_rightIdx, d_FinalLTLIdx
                );
            }

        }

        else {

            UINT_64 CS[maxNumOfTraces];
            applyOperator(op, CS, d_LTLcache, ldx, rdx, numOfTraces);

            if (onTheFly) {
                processOnTheFly(
                    CS, tid, numOfTraces, numOfP, ldx, rdx,
                    d_temp_leftIdx, d_temp_rightIdx, d_FinalLTLIdx
                );
            }

            else {
                bool CS_is_unique =
                    processUniqueCS(CS, numOfTraces, RlxUnqChkTyp, lenSum, cHashSet, iHashSet);
                insertInCache(
                    CS_is_unique, CS, tid, numOfTraces, numOfP, ldx, rdx,
                    d_temp_LTLcache, d_temp_leftIdx, d_temp_rightIdx, d_FinalLTLIdx
                );
            }

        }

    }

}

// Generating the final RE string recursively
// When all the left and right indices are ready in the host
std::string toString(
    int index,
    std::map<int, std::pair<int, int>>& indicesMap,
    const std::set<char>& alphabet,
    const int* startPoints)
{

    if (index < alphabet.size()) {
        std::string s(1, *next(alphabet.begin(), index));
        return s;
    }
    int i = 0;
    while (index >= startPoints[i]) { i++; }
    i--;

    if (i % 7 == 0) {
        std::string res = toString(indicesMap[index].first, indicesMap, alphabet, startPoints);
        return "~(" + res + ")";
    }

    if (i % 7 == 1) {
        std::string left = toString(indicesMap[index].first, indicesMap, alphabet, startPoints);
        std::string right = toString(indicesMap[index].second, indicesMap, alphabet, startPoints);
        return "(" + left + ")" + "&" + "(" + right + ")";
    }

    if (i % 7 == 2) {
        std::string left = toString(indicesMap[index].first, indicesMap, alphabet, startPoints);
        std::string right = toString(indicesMap[index].second, indicesMap, alphabet, startPoints);
        return "(" + left + ")" + "|" + "(" + right + ")";
    }

    if (i % 7 == 3) {
        std::string res = toString(indicesMap[index].first, indicesMap, alphabet, startPoints);
        return "X(" + res + ")";
    }

    if (i % 7 == 4) {
        std::string res = toString(indicesMap[index].first, indicesMap, alphabet, startPoints);
        return "F(" + res + ")";
    }

    if (i % 7 == 5) {
        std::string res = toString(indicesMap[index].first, indicesMap, alphabet, startPoints);
        return "G(" + res + ")";
    }

    std::string left = toString(indicesMap[index].first, indicesMap, alphabet, startPoints);
    std::string right = toString(indicesMap[index].second, indicesMap, alphabet, startPoints);
    return "(" + left + ")" + "U" + "(" + right + ")";

}

// Bringing the left and right indices of the LTL from device to host
// If LTL is found, this index is from the temp memory               (temp = true)
// For printing other LTLs if needed, indices are in the main memory (temp = false)
std::string LTLtoString(
    bool temp,
    const int FinalLTLIdx,
    const int lastIdx,
    const std::set<char>& alphabet,
    const int* startPoints,
    const int* d_leftIdx,
    const int* d_rightIdx,
    const int* d_temp_leftIdx,
    const int* d_temp_rightIdx)
{

    auto* LIdx = new int[1];
    auto* RIdx = new int[1];

    if (temp) {
        checkCuda(hipMemcpy(LIdx, d_temp_leftIdx + FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost));
        checkCuda(hipMemcpy(RIdx, d_temp_rightIdx + FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost));
    }
    else {
        checkCuda(hipMemcpy(LIdx, d_leftIdx + FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost));
        checkCuda(hipMemcpy(RIdx, d_rightIdx + FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost));
    }

    auto alphabetSize = static_cast<int> (alphabet.size());

    int* d_resIndices;
    checkCuda(hipMalloc(&d_resIndices, 600 * sizeof(int)));

    thrust::device_ptr<int> d_resIndices_ptr(d_resIndices);
    thrust::fill(d_resIndices_ptr, d_resIndices_ptr + 600, -1);

    if (*LIdx >= alphabetSize) generateResIndices << <1, 1 >> > (*LIdx, alphabetSize, d_leftIdx, d_rightIdx, d_resIndices);
    if (*RIdx >= alphabetSize) generateResIndices << <1, 1 >> > (*RIdx, alphabetSize, d_leftIdx, d_rightIdx, d_resIndices);

    int resIndices[600];
    checkCuda(hipMemcpy(resIndices, d_resIndices, 600 * sizeof(int), hipMemcpyDeviceToHost));

    std::map<int, std::pair<int, int>> indicesMap;

    if (temp) indicesMap.insert(std::make_pair(INT_MAX - 1, std::make_pair(*LIdx, *RIdx)));
    else      indicesMap.insert(std::make_pair(FinalLTLIdx, std::make_pair(*LIdx, *RIdx)));

    int i = 0;
    while (resIndices[i] != -1 && i + 2 < 600) {
        int ltl = resIndices[i];
        int l = resIndices[i + 1];
        int r = resIndices[i + 2];
        indicesMap.insert(std::make_pair(ltl, std::make_pair(l, r)));
        i += 3;
    }

    if (i + 2 >= 600) return "Size of the output is too big";

    hipFree(d_resIndices);

    if (temp) return toString(INT_MAX - 1, indicesMap, alphabet, startPoints);
    else      return toString(FinalLTLIdx, indicesMap, alphabet, startPoints);

}

// Transfering the unique CSs from temp to main LTLcache
void storeUniqueLTLs(
    int N,
    int& lastIdx,
    const int numOfTraces,
    const int LTLcacheCapacity,
    bool& onTheFly,
    UINT_64* d_LTLcache,
    UINT_64* d_temp_LTLcache,
    int* d_leftIdx,
    int* d_rightIdx,
    int* d_temp_leftIdx,
    int* d_temp_rightIdx)
{

    thrust::device_ptr<UINT_64> new_end_ptr;
    thrust::device_ptr<UINT_64> d_LTLcache_ptr(d_LTLcache + numOfTraces * lastIdx);
    thrust::device_ptr<UINT_64> d_temp_LTLcache_ptr(d_temp_LTLcache);
    thrust::device_ptr<int> d_leftIdx_ptr(d_leftIdx + lastIdx);
    thrust::device_ptr<int> d_rightIdx_ptr(d_rightIdx + lastIdx);
    thrust::device_ptr<int> d_temp_leftIdx_ptr(d_temp_leftIdx);
    thrust::device_ptr<int> d_temp_rightIdx_ptr(d_temp_rightIdx);

    new_end_ptr = // end of d_temp_LTLcache
        thrust::remove(d_temp_LTLcache_ptr, d_temp_LTLcache_ptr + numOfTraces * N, (UINT_64)-1);
    thrust::remove(d_temp_leftIdx_ptr, d_temp_leftIdx_ptr + N, -1);
    thrust::remove(d_temp_rightIdx_ptr, d_temp_rightIdx_ptr + N, -1);

    // It stores all (or a part of) unique CSs until language cahce gets full
    // If language cache gets full, it makes onTheFly mode on
    int numberOfNewUniqueLTLs = static_cast<int>(new_end_ptr - d_temp_LTLcache_ptr) / numOfTraces;
    if (lastIdx + numberOfNewUniqueLTLs > LTLcacheCapacity) {
        N = LTLcacheCapacity - lastIdx;
        onTheFly = true;
    }
    else N = numberOfNewUniqueLTLs;

    thrust::copy_n(d_temp_LTLcache_ptr, numOfTraces * N, d_LTLcache_ptr);
    thrust::copy_n(d_temp_leftIdx_ptr, N, d_leftIdx_ptr);
    thrust::copy_n(d_temp_rightIdx_ptr, N, d_rightIdx_ptr);

    lastIdx += N;

}

int costOf(const int index, const int* startPoints) {
    int i = 0;
    while (index >= startPoints[i]) { i++; }
    return((i - 1) / 4);
}

std::string LTLI(
    const unsigned short* costFun,
    const unsigned short maxCost,
    const unsigned int RlxUnqChkTyp,
    const unsigned int NegType,
    const std::set<char> alphabet,
    int& LTLcost,
    std::uint64_t& allLTLs,
    const std::vector<std::vector<std::string>> pos,
    const std::vector<std::vector<std::string>> neg) {

    // --------------------------------
    // Generating and checking alphabet
    // --------------------------------

    const int numOfP = pos.size();
    const int numOfN = neg.size();
    const int numOfTraces = numOfP + numOfN;

    int maxLenOfTraces{};
    auto* traceLen = new char[numOfTraces];

    int TLIdx{};
    int lenSum{};
    for (const auto& trace : pos) {
        lenSum += trace.size();
        traceLen[TLIdx++] = trace.size();
        if (trace.size() > maxLenOfTraces) maxLenOfTraces = trace.size();
    }
    for (const auto& trace : neg) {
        lenSum += trace.size();
        traceLen[TLIdx++] = trace.size();
        if (trace.size() > maxLenOfTraces) maxLenOfTraces = trace.size();
    }

    if (numOfTraces > maxNumOfTraces || maxLenOfTraces > sizeof(UINT_64) * 8 - 1) {
        printf("In this version, The input can have:\n");
        printf("1) At most %zu traces. It is currently %d.\n", maxNumOfTraces, numOfTraces);
        printf("2) Max(len(trace)) = %d. It is currently %d.\n", static_cast<int>(sizeof(UINT_64) * 8 - 1), maxLenOfTraces);
        return "see_the_error";
    }

    // Copying the length of traces into the constant memory
    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_traceLen), traceLen, numOfTraces * sizeof(char)));

    const int alphabetSize = static_cast<int>(alphabet.size());

    auto* LTLcache = new UINT_64[alphabetSize * numOfTraces];

    // Index of the last free position in the LTLcache
    int lastIdx{};

#ifndef MEASUREMENT_MODE
    printf("Cost %-2d | (A) | AllLTLs: %-11lu | StoredLTLs: %-10d | ToBeChecked: %-10d \n",
        costFun[0], allLTLs, 0, alphabetSize);
#endif

    int index{};
    for (int i = 0; i < alphabetSize; ++i) {
        bool found = true;
        std::string ch(1, *next(alphabet.begin(), i));
        for (const auto& trace : pos) {
            UINT_64 binTrace{};
            UINT_64 idx = 1;
            for (const auto& token : trace) {
                for (const auto& c : token) {
                    if (c == ch[0]) {
                        binTrace |= idx;
                        break;
                    }
                }
                idx <<= 1;
            }
            LTLcache[index++] = binTrace;
            if (!(binTrace & 1)) found = false;
        }
        for (const auto& trace : neg) {
            UINT_64 binTrace{};
            UINT_64 idx = 1;
            for (const auto& token : trace) {
                for (const auto& c : token) {
                    if (c == ch[0]) {
                        binTrace |= idx;
                        break;
                    }
                }
                idx <<= 1;
            }
            LTLcache[index++] = binTrace;
            if (binTrace & 1) found = false;
        }
        allLTLs++; lastIdx++;
        if (found) return ch;
    }

    // ------------------------------------
    // Memory allocation and initialisation
    // ------------------------------------

    // cost function
    int c1 = costFun[0]; // cost of p
    int c2 = costFun[1]; // cost of ~
    int c3 = costFun[2]; // cost of &
    int c4 = costFun[3]; // cost of |
    int c5 = costFun[4]; // cost of X
    int c6 = costFun[5]; // cost of F
    int c7 = costFun[6]; // cost of G
    int c8 = costFun[7]; // cost of U

    int maxAllocationSize;
    hipDeviceGetAttribute(&maxAllocationSize, hipDeviceAttributeMaxPitch, 0);

    const int LTLcacheCapacity = maxAllocationSize / (numOfTraces * sizeof(UINT_64)) * 1.5;
    const int temp_LTLcacheCapacity = LTLcacheCapacity / 2;

    // const int LTLcacheCapacity = 2000000;
    // const int temp_LTLcacheCapacity = 100000000;

    // 7 for ~, &, |, X, F, G, U
    int* startPoints = new int[(maxCost + 2) * 7]();
    startPoints[c1 * 7 + 6] = lastIdx;
    startPoints[(c1 + 1) * 7] = lastIdx;

    int* d_FinalLTLIdx;
    auto* FinalLTLIdx = new int[1]; *FinalLTLIdx = -1;
    checkCuda(hipMalloc(&d_FinalLTLIdx, sizeof(int)));
    checkCuda(hipMemcpy(d_FinalLTLIdx, FinalLTLIdx, sizeof(int), hipMemcpyHostToDevice));

    UINT_64* d_LTLcache, * d_temp_LTLcache;
    int* d_leftIdx, * d_rightIdx, * d_temp_leftIdx, * d_temp_rightIdx;
    checkCuda(hipMalloc(&d_leftIdx, LTLcacheCapacity * sizeof(int)));
    checkCuda(hipMalloc(&d_rightIdx, LTLcacheCapacity * sizeof(int)));
    checkCuda(hipMalloc(&d_temp_leftIdx, temp_LTLcacheCapacity * sizeof(int)));
    checkCuda(hipMalloc(&d_temp_rightIdx, temp_LTLcacheCapacity * sizeof(int)));
    checkCuda(hipMalloc(&d_LTLcache, LTLcacheCapacity * numOfTraces * sizeof(UINT_64)));
    checkCuda(hipMalloc(&d_temp_LTLcache, temp_LTLcacheCapacity * numOfTraces * sizeof(UINT_64)));

    using hash_set_t = warpcore::HashSet<
        UINT_64,         // key type
        UINT_64(0) - 1,  // empty key
        UINT_64(0) - 2,  // tombstone key
        warpcore::probing_schemes::QuadraticProbing<warpcore::hashers::MurmurHash <UINT_64>>>;

    hash_set_t cHashSet(2 * LTLcacheCapacity);
    hash_set_t iHashSet(2 * LTLcacheCapacity);

    checkCuda(hipMemcpy(d_LTLcache, LTLcache, alphabetSize * numOfTraces * sizeof(UINT_64), hipMemcpyHostToDevice));
    hashSetsInitialisation<hash_set_t> << <1, alphabetSize >> > (numOfTraces, RlxUnqChkTyp, lenSum, cHashSet, iHashSet, d_LTLcache);

    // ----------------------------
    // Enumeration of the next LTLs
    // ----------------------------

    bool onTheFly = false, lastRound = false;
    int shortageCost = -1;

    for (LTLcost = c1 + 1; LTLcost <= maxCost; ++LTLcost) {


        // Once it uses a previous cost that is not fully stored, it should continue as the last round
        if (onTheFly) {
            int dif = LTLcost - shortageCost;
            if (dif == c2 || dif == c1 + c3 || dif == c1 + c4 || dif == c5 || dif == c6 || dif == c7 || dif == c1 + c8) lastRound = true;
        }


        // negation (~)
        // NegType = 1 is for negation of phi
        // NegType = 2 is for negation of char only
        if ((NegType == 1 && LTLcost - c2 >= c1) || (NegType == 2 && LTLcost - c2 == c1))
            if (LTLcost - c2 >= c1) {

                int Idx1 = startPoints[(LTLcost - c2) * 7];
                int Idx2 = startPoints[(LTLcost - c2 + 1) * 7] - 1;
                int N = Idx2 - Idx1 + 1;

                if (N) {
                    int x = Idx1, y;
                    do {
                        y = x + std::min(temp_LTLcacheCapacity - 1, Idx2 - x);
                        N = (y - x + 1);
#ifndef MEASUREMENT_MODE
                        printf("Cost %-2d | (~) | AllLTLs: %-11lu | StoredLTLs: %-10d | ToBeChecked: %-10d \n",
                            LTLcost, allLTLs, lastIdx, N);
#endif
                        int Blc = (N + 1023) / 1024;
                        processOperator<hash_set_t> << <Blc, 1024 >> > (
                            '~', x, y, 0, 0, numOfP, numOfN, RlxUnqChkTyp, lenSum, onTheFly,
                            d_LTLcache, d_temp_LTLcache, d_temp_leftIdx, d_temp_rightIdx,
                            cHashSet, iHashSet, d_FinalLTLIdx
                            );
                        checkCuda(hipPeekAtLastError());
                        checkCuda(hipMemcpy(FinalLTLIdx, d_FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost));
                        allLTLs += N;
                        if (*FinalLTLIdx != -1) { startPoints[LTLcost * 7 + 1] = INT_MAX; goto exitEnumeration; }
                        if (!onTheFly) storeUniqueLTLs(N, lastIdx, numOfTraces, LTLcacheCapacity, onTheFly, d_LTLcache, d_temp_LTLcache,
                            d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);
                        x = y + 1;
                    } while (y < Idx2);
                }

            }
        startPoints[LTLcost * 7 + 1] = lastIdx;

        // intersection (&)
        for (int i = c1; 2 * i <= LTLcost - c3; ++i) {

            int Idx1 = startPoints[i * 7];
            int Idx2 = startPoints[(i + 1) * 7] - 1;
            int Idx3 = startPoints[(LTLcost - i - c3) * 7];
            int Idx4 = startPoints[(LTLcost - i - c3 + 1) * 7] - 1;
            int N = (Idx4 - Idx3 + 1) * (Idx2 - Idx1 + 1);

            if (N) {
                int x = Idx3, y;
                do {
                    y = x + std::min(temp_LTLcacheCapacity / (Idx2 - Idx1 + 1) - 1, Idx4 - x);
                    N = (y - x + 1) * (Idx2 - Idx1 + 1);
#ifndef MEASUREMENT_MODE
                    printf("Cost %-2d | (&) | AllLTLs: %-11lu | StoredLTLs: %-10d | ToBeChecked: %-10d \n",
                        LTLcost, allLTLs, lastIdx, N);
#endif
                    int Blc = (N + 1023) / 1024;
                    processOperator<hash_set_t> << <Blc, 1024 >> > (
                        '&', Idx1, Idx2, x, y, numOfP, numOfN, RlxUnqChkTyp, lenSum, onTheFly,
                        d_LTLcache, d_temp_LTLcache, d_temp_leftIdx, d_temp_rightIdx,
                        cHashSet, iHashSet, d_FinalLTLIdx
                        );
                    checkCuda(hipPeekAtLastError());
                    checkCuda(hipMemcpy(FinalLTLIdx, d_FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost));
                    allLTLs += N;
                    if (*FinalLTLIdx != -1) { startPoints[LTLcost * 7 + 2] = INT_MAX; goto exitEnumeration; }
                    if (!onTheFly) storeUniqueLTLs(N, lastIdx, numOfTraces, LTLcacheCapacity, onTheFly, d_LTLcache, d_temp_LTLcache,
                        d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);
                    x = y + 1;
                } while (y < Idx4);
            }

        }
        startPoints[LTLcost * 7 + 2] = lastIdx;

        // union (|)
        for (int i = c1; 2 * i <= LTLcost - c4; ++i) {

            int Idx1 = startPoints[i * 7];
            int Idx2 = startPoints[(i + 1) * 7] - 1;
            int Idx3 = startPoints[(LTLcost - i - c4) * 7];
            int Idx4 = startPoints[(LTLcost - i - c4 + 1) * 7] - 1;
            int N = (Idx4 - Idx3 + 1) * (Idx2 - Idx1 + 1);

            if (N) {
                int x = Idx3, y;
                do {
                    y = x + std::min(temp_LTLcacheCapacity / (Idx2 - Idx1 + 1) - 1, Idx4 - x);
                    N = (y - x + 1) * (Idx2 - Idx1 + 1);
#ifndef MEASUREMENT_MODE
                    printf("Cost %-2d | (|) | AllLTLs: %-11lu | StoredLTLs: %-10d | ToBeChecked: %-10d \n",
                        LTLcost, allLTLs, lastIdx, N);
#endif
                    int Blc = (N + 1023) / 1024;
                    processOperator<hash_set_t> << <Blc, 1024 >> > (
                        '|', Idx1, Idx2, x, y, numOfP, numOfN, RlxUnqChkTyp, lenSum, onTheFly,
                        d_LTLcache, d_temp_LTLcache, d_temp_leftIdx, d_temp_rightIdx,
                        cHashSet, iHashSet, d_FinalLTLIdx
                        );
                    checkCuda(hipPeekAtLastError());
                    checkCuda(hipMemcpy(FinalLTLIdx, d_FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost));
                    allLTLs += N;
                    if (*FinalLTLIdx != -1) { startPoints[LTLcost * 7 + 3] = INT_MAX; goto exitEnumeration; }
                    if (!onTheFly) storeUniqueLTLs(N, lastIdx, numOfTraces, LTLcacheCapacity, onTheFly, d_LTLcache, d_temp_LTLcache,
                        d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);
                    x = y + 1;
                } while (y < Idx4);
            }

        }
        startPoints[LTLcost * 7 + 3] = lastIdx;

        // next (X)
        if (LTLcost - c5 >= c1) {

            int Idx1 = startPoints[(LTLcost - c5) * 7];
            int Idx2 = startPoints[(LTLcost - c5 + 1) * 7] - 1;
            int N = Idx2 - Idx1 + 1;

            if (N) {
                int x = Idx1, y;
                do {
                    y = x + std::min(temp_LTLcacheCapacity - 1, Idx2 - x);
                    N = (y - x + 1);
#ifndef MEASUREMENT_MODE
                    printf("Cost %-2d | (X) | AllLTLs: %-11lu | StoredLTLs: %-10d | ToBeChecked: %-10d \n",
                        LTLcost, allLTLs, lastIdx, N);
#endif
                    int Blc = (N + 1023) / 1024;
                    processOperator<hash_set_t> << <Blc, 1024 >> > (
                        'X', x, y, 0, 0, numOfP, numOfN, RlxUnqChkTyp, lenSum, onTheFly,
                        d_LTLcache, d_temp_LTLcache, d_temp_leftIdx, d_temp_rightIdx,
                        cHashSet, iHashSet, d_FinalLTLIdx
                        );
                    checkCuda(hipPeekAtLastError());
                    checkCuda(hipMemcpy(FinalLTLIdx, d_FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost));
                    allLTLs += N;
                    if (*FinalLTLIdx != -1) { startPoints[LTLcost * 7 + 4] = INT_MAX; goto exitEnumeration; }
                    if (!onTheFly) storeUniqueLTLs(N, lastIdx, numOfTraces, LTLcacheCapacity, onTheFly, d_LTLcache, d_temp_LTLcache,
                        d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);
                    x = y + 1;
                } while (y < Idx2);
            }

        }
        startPoints[LTLcost * 7 + 4] = lastIdx;

        // finally (F)
        if (LTLcost - c6 >= c1) {

            int Idx1 = startPoints[(LTLcost - c6) * 7];
            int Idx2 = startPoints[(LTLcost - c6 + 1) * 7] - 1;
            int N = Idx2 - Idx1 + 1;

            if (N) {
                int x = Idx1, y;
                do {
                    y = x + std::min(temp_LTLcacheCapacity - 1, Idx2 - x);
                    N = (y - x + 1);
#ifndef MEASUREMENT_MODE
                    printf("Cost %-2d | (F) | AllLTLs: %-11lu | StoredLTLs: %-10d | ToBeChecked: %-10d \n",
                        LTLcost, allLTLs, lastIdx, N);
#endif
                    int Blc = (N + 1023) / 1024;
                    processOperator<hash_set_t> << <Blc, 1024 >> > (
                        'F', x, y, 0, 0, numOfP, numOfN, RlxUnqChkTyp, lenSum, onTheFly,
                        d_LTLcache, d_temp_LTLcache, d_temp_leftIdx, d_temp_rightIdx,
                        cHashSet, iHashSet, d_FinalLTLIdx
                        );
                    checkCuda(hipPeekAtLastError());
                    checkCuda(hipMemcpy(FinalLTLIdx, d_FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost));
                    allLTLs += N;
                    if (*FinalLTLIdx != -1) { startPoints[LTLcost * 7 + 5] = INT_MAX; goto exitEnumeration; }
                    if (!onTheFly) storeUniqueLTLs(N, lastIdx, numOfTraces, LTLcacheCapacity, onTheFly, d_LTLcache, d_temp_LTLcache,
                        d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);
                    x = y + 1;
                } while (y < Idx2);
            }

        }
        startPoints[LTLcost * 7 + 5] = lastIdx;

        // globally (G)
        if (LTLcost - c7 >= c1) {

            int Idx1 = startPoints[(LTLcost - c7) * 7];
            int Idx2 = startPoints[(LTLcost - c7 + 1) * 7] - 1;
            int N = Idx2 - Idx1 + 1;

            if (N) {
                int x = Idx1, y;
                do {
                    y = x + std::min(temp_LTLcacheCapacity - 1, Idx2 - x);
                    N = (y - x + 1);
#ifndef MEASUREMENT_MODE
                    printf("Cost %-2d | (G) | AllLTLs: %-11lu | StoredLTLs: %-10d | ToBeChecked: %-10d \n",
                        LTLcost, allLTLs, lastIdx, N);
#endif
                    int Blc = (N + 1023) / 1024;
                    processOperator<hash_set_t> << <Blc, 1024 >> > (
                        'G', x, y, 0, 0, numOfP, numOfN, RlxUnqChkTyp, lenSum, onTheFly,
                        d_LTLcache, d_temp_LTLcache, d_temp_leftIdx, d_temp_rightIdx,
                        cHashSet, iHashSet, d_FinalLTLIdx
                        );
                    checkCuda(hipPeekAtLastError());
                    checkCuda(hipMemcpy(FinalLTLIdx, d_FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost));
                    allLTLs += N;
                    if (*FinalLTLIdx != -1) { startPoints[LTLcost * 7 + 6] = INT_MAX; goto exitEnumeration; }
                    if (!onTheFly) storeUniqueLTLs(N, lastIdx, numOfTraces, LTLcacheCapacity, onTheFly, d_LTLcache, d_temp_LTLcache,
                        d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);
                    x = y + 1;
                } while (y < Idx2);
            }

        }
        startPoints[LTLcost * 7 + 6] = lastIdx;

        // until (U)
        for (int i = c1; 2 * i <= LTLcost - c8; ++i) {

            int Idx1 = startPoints[i * 7];
            int Idx2 = startPoints[(i + 1) * 7] - 1;
            int Idx3 = startPoints[(LTLcost - i - c8) * 7];
            int Idx4 = startPoints[(LTLcost - i - c8 + 1) * 7] - 1;
            int N = (Idx4 - Idx3 + 1) * (Idx2 - Idx1 + 1);

            if (N) {
                int x = Idx3, y;
                do {
                    y = x + std::min(temp_LTLcacheCapacity / (2 * (Idx2 - Idx1 + 1)) - 1, Idx4 - x); // 2 is for until only (lUr and rUl)
                    N = (y - x + 1) * (Idx2 - Idx1 + 1);
#ifndef MEASUREMENT_MODE
                    printf("Cost %-2d | (U) | AllLTLs: %-11lu | StoredLTLs: %-10d | ToBeChecked: %-10d \n",
                        LTLcost, allLTLs, lastIdx, 2 * N);
#endif
                    int Blc = (N + 1023) / 1024;
                    processOperator<hash_set_t> << <Blc, 1024 >> > (
                        'U', Idx1, Idx2, x, y, numOfP, numOfN, RlxUnqChkTyp, lenSum, onTheFly,
                        d_LTLcache, d_temp_LTLcache, d_temp_leftIdx, d_temp_rightIdx,
                        cHashSet, iHashSet, d_FinalLTLIdx
                        );
                    checkCuda(hipPeekAtLastError());
                    checkCuda(hipMemcpy(FinalLTLIdx, d_FinalLTLIdx, sizeof(int), hipMemcpyDeviceToHost));
                    allLTLs += 2 * N;
                    if (*FinalLTLIdx != -1) { startPoints[(LTLcost + 1) * 7] = INT_MAX; goto exitEnumeration; }
                    if (!onTheFly) storeUniqueLTLs(2 * N, lastIdx, numOfTraces, LTLcacheCapacity, onTheFly, d_LTLcache, d_temp_LTLcache,
                        d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);
                    x = y + 1;
                } while (y < Idx4);
            }

        }
        startPoints[(LTLcost + 1) * 7] = lastIdx;

        if (lastRound) break;
        if (onTheFly && shortageCost == -1) shortageCost = LTLcost;

    }

    if (LTLcost == maxCost + 1) LTLcost--;

exitEnumeration:

    std::string output;
    bool isLTLFromTempLTLcache = true;

    if (*FinalLTLIdx != -1) {

        output = LTLtoString(isLTLFromTempLTLcache, *FinalLTLIdx, lastIdx, alphabet, startPoints,
            d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);

    }
    else {

        output = "not_found";

    }

    /*
    const int NNN = 1000;
    auto *cache2 = new UINT_64[numOfTraces * NNN];
    auto *left_indices = new int[NNN];
    auto *right_indices = new int[NNN];
    checkCuda( hipMemcpy(cache2, d_LTLcache, numOfTraces * NNN * sizeof(UINT_64), hipMemcpyDeviceToHost) );
    checkCuda( hipMemcpy(left_indices, d_leftIdx, NNN * sizeof(int), hipMemcpyDeviceToHost) );
    checkCuda( hipMemcpy(right_indices, d_rightIdx, NNN * sizeof(int), hipMemcpyDeviceToHost) );

    for (int i = 0; i < NNN; ++i) {
        std::cout << i << " --> \t \t";
        std::string out = LTLtoString(false, i, lastIdx, alphabet, startPoints,
        d_leftIdx, d_rightIdx, d_temp_leftIdx, d_temp_rightIdx);
        std::cout << out << "\t \t";
        for (int j = 0; j < numOfTraces; ++j) {
            std::cout << cache2[i * numOfTraces + j] << "\t";
        }
        std::cout << std::endl;
    }
    */

    // cleanup
    hipFree(d_LTLcache);
    hipFree(d_FinalLTLIdx);
    hipFree(d_temp_LTLcache);
    hipFree(d_leftIdx);
    hipFree(d_rightIdx);
    hipFree(d_temp_leftIdx);
    hipFree(d_temp_rightIdx);

    return output;

}

// Reading the input file
bool readFile(
    const std::string& fileName,
    std::set<char>& alphabet,
    std::vector<std::vector<std::string>>& pos,
    std::vector<std::vector<std::string>>& neg)
{

    std::ifstream file(fileName);
    if (file.is_open()) {
        file.seekg(0, std::ios::end);
        std::string line;
        char ch;
        bool foundNewline = false;
        while (!foundNewline && file.tellg() > 0) {
            file.seekg(-2, std::ios::cur);
            file.get(ch);
            if (ch == '\n') foundNewline = true;
        }
        std::getline(file, line);
        std::string alpha;
        for (auto& c : line)
            if (c >= 'a' && c <= 'z') {
                alphabet.insert(c);
                alpha += c;
            }
        file.seekg(0, std::ios::beg);
        while (std::getline(file, line)) {
            std::vector<std::string> trace;
            if (line != "---") {
                std::string token;
                int j{};
                for (auto& c : line) {
                    if (c == ';') {
                        trace.push_back(token);
                        token = "";
                        j = 0;
                    }
                    else if (c == ',') continue;
                    else {
                        if (c == '1') token += alpha[j];
                        j++;
                    }
                }
                trace.push_back(token);
                pos.push_back(trace);
            }
            else break;
        }
        while (std::getline(file, line)) {
            std::vector<std::string> trace;
            if (line != "---") {
                std::string token;
                int j{};
                for (auto& c : line) {
                    if (c == ';') {
                        trace.push_back(token);
                        token = "";
                        j = 0;
                    }
                    else if (c == ',') continue;
                    else {
                        if (c == '1') token += alpha[j];
                        j++;
                    }
                }
                trace.push_back(token);
                neg.push_back(trace);
            }
            else break;
        }
        file.close();
        return true;
    }
    else printf("Failed to open the input file.\n");

    return false;

}

int main(int argc, char* argv[]) {

    // -----------------
    // Reading the input
    // -----------------

    if (argc != 13) {
        printf("Arguments should be in the form of\n");
        printf("-----------------------------------------------------------------\n");
        printf("%s <input_file_address> <c1> <c2> <c3> <c4> <c5> <c6> <c7> <c8> <maxCost> <RlxUnqChkTyp> <NegType>\n", argv[0]);
        printf("-----------------------------------------------------------------\n");
        printf("\nFor example\n");
        printf("-----------------------------------------------------------------\n");
        printf("%s ./input.txt 1 1 1 1 1 1 1 1 500 3 2\n", argv[0]);
        printf("-----------------------------------------------------------------\n");
        return 0;
    }

    bool argError = false;
    for (int i = 2; i < argc - 2; ++i) {
        if (std::atoi(argv[i]) <= 0 || std::atoi(argv[i]) > SHRT_MAX) {
            printf("Argument number %d, \"%s\", should be a positive short integer.\n", i, argv[i]);
            argError = true;
        }
    }
    if (std::atoi(argv[11]) < 1 || std::atoi(argv[11]) > 3) {
        printf("Argument number 11, RlxUnqChkTyp = \"%s\", should be 1, 2, or 3.\n", argv[11]);
        argError = true;
    }
    if (std::atoi(argv[12]) < 1 || std::atoi(argv[12]) > 2) {
        printf("Argument number 12, NegType = \"%s\", should be 1, or 2.\n", argv[12]);
        argError = true;
    }

    if (argError) return 0;

    std::string fileName = argv[1];
    std::set<char> alphabet;
    std::vector<std::vector<std::string>> pos, neg;
    if (!readFile(fileName, alphabet, pos, neg)) return 0;
    unsigned short costFun[8];
    for (int i = 0; i < 8; i++)
        costFun[i] = std::atoi(argv[i + 2]);
    unsigned short maxCost = std::atoi(argv[10]);
    unsigned int RlxUnqChkTyp = std::atoi(argv[11]);
    unsigned int NegType = std::atoi(argv[12]);

    // --------------------------------------
    // Linear Temporal Logic Inference (LTLI)
    // --------------------------------------

#ifdef MEASUREMENT_MODE
    auto start = std::chrono::high_resolution_clock::now();
#endif

    std::uint64_t allLTLs{}; int LTLcost = costFun[0];
    std::string output = LTLI(costFun, maxCost, RlxUnqChkTyp, NegType, alphabet, LTLcost, allLTLs, pos, neg);
    if (output == "see_the_error") return 0;

#ifdef MEASUREMENT_MODE
    auto stop = std::chrono::high_resolution_clock::now();
#endif

    // -------------------
    // Printing the output
    // -------------------

    printf("\nPositive: \n");
    for (const auto& trace : pos) {
        printf("\t");
        for (const auto& t : trace) {
            std::string s;
            for (const auto& c : t) {
                s += c; s += ", ";
            }
            printf("{%s}\t", s.substr(0, s.length() - 2).c_str());
        }
        printf("\n");
    }

    printf("\nNegative: \n");
    for (const auto& trace : neg) {
        printf("\t");
        for (const auto& t : trace) {
            std::string s;
            for (const auto& c : t) {
                s += c; s += ", ";
            }
            printf("{%s}\t", s.substr(0, s.length() - 2).c_str());
        }
        printf("\n");
    }

    printf("\nCost Function: p:%u, ~:%u, &:%u, |:%u, X:%u, F:%u, G:%u, U:%u",
        costFun[0], costFun[1], costFun[2], costFun[3], costFun[4], costFun[5], costFun[6], costFun[7]);
    printf("\nNumber of Traces: %d", static_cast<int>(pos.size() + neg.size()));
#ifdef MEASUREMENT_MODE
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
    printf("\nNumber of All LTLs: %lu", allLTLs);
    printf("\nCost of Final LTL: %d", LTLcost);
    printf("\nRunning Time: %f s", (double)duration * 0.000001);
#endif
    printf("\n\nLTL: \"%s\"\n", output.c_str());

    return 0;

}